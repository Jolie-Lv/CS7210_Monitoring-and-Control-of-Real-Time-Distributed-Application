#include "hip/hip_runtime.h"
#include "Jammer.h"


// /usr/include/nvidia/gdk/



int main(int argc, char const *argv[])
{
    start((argc > 1 ) ? argv[1] : "Jammer1");
    
    
    while(runing) {
        cout<<"Waiting on DRFM Message every second..."<<endl;
        ProtoMessage msg = receiveProtobufMessage(drfmSocket);
        if(msg.dest_device() == id){
            if(msg.signal() == ProtoMessage::TERMINATE){
                terminator(msg);
            } else if (msg.signal() == ProtoMessage::RESET){
                resetDevice(msg);
            } else if (msg.has_sensor()) {
                ProtoMessage::SensorMessage smsg = msg.sensor();
                load = smsg.workload();
            }
        } 
        if(runing) {
            
            for (int i = 0; i < load; ++i)
            {
                loadGPU();
            }
            std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();
            get_mon_data_GPU(&mon_data);
            get_mon_data_CPU(&mon_data);
            get_mon_data_MEM(&mon_data);
            ProtoMessage sendMsg;
            sendMsg.set_dest_device(id);
            sendMsg.set_signal(ProtoMessage::DEFAULT);
            ProtoMessage::JammerMessage *jmsg = sendMsg.mutable_jammer();
            jmsg->set_performacnce(getTimeSinceLastNano(start));
            jmsg->set_gpuutil(mon_data.cpu_util);
            jmsg->set_cpumemutil(mon_data.mem);
            jmsg->set_gpumemutil(mon_data.utilGPU_mem);
            jmsg->set_gpuutil(mon_data.utilGPU_cpu);
            jmsg->set_workload(load);
            jmsg->set_app(mon_data.app);
            sendMsg.set_time(getCurrrentimeInMicroseconds());
            sendProtobufMessage(drfmSocket,sendMsg);
            cout <<"\t\tCPU : "<<mon_data.cpu_util<<endl;
            cout <<"\t\tMemory :"<<mon_data.mem<<endl;
        }
    }
	google::protobuf::ShutdownProtobufLibrary();
	return 0;
}

void start(const char * idc){
    
    id = string(idc);
    resetFilename = id+"reset";
    runing = true;
    load = 1;

    FILE* file;
    struct tms timeSample;
    char line[128];
    lastCPU = times(&timeSample);
    lastSysCPU = timeSample.tms_stime;
    lastUserCPU = timeSample.tms_utime;

    file = fopen("/proc/cpuinfo", "r");
    numProcessors = 0;
    while(fgets(line, 128, file) != NULL){
        if (strncmp(line, "processor", 9) == 0) numProcessors++;
    }
    fclose(file);

    // Connect DRFM();

    struct addrinfo hints;
    memset(&hints, 0, sizeof(struct addrinfo));
    hints.ai_family = AF_INET;    /* Allow IPv4 or IPv6 */
    hints.ai_socktype = SOCK_STREAM; 
    struct addrinfo *result,*rp;   
    getaddrinfo(name2hostname["DRFM"].c_str(),
                Name2port[id].c_str(),
                &hints,
                &result);

    for (rp = result; rp != NULL; rp = rp->ai_next) {
        drfmSocket = socket(rp->ai_family, rp->ai_socktype,
                     rp->ai_protocol);
        if (drfmSocket == -1) continue;
            struct timeval t;    
            t.tv_sec = 1;
            t.tv_usec = 0;
            setsockopt(
                drfmSocket,     // Socket descriptor
                SOL_SOCKET, // To manipulate options at the sockets API level
                SO_RCVTIMEO,// Specify the receiving or sending timeouts 
                (const void *)(&t), // option values
                sizeof(t) 
            );
        if (connect(drfmSocket, rp->ai_addr, rp->ai_addrlen) != -1) {
            cout << "Connected to DRFM" << endl;
            break;                  /* Success */
       }
       close(drfmSocket);
    }
    if (rp == NULL) {
        cerr <<"Failed to connect to DRFM\n";
        exit(1);
    }
    freeaddrinfo(result);
    
    ProtoMessage msg;
    msg.set_dest_device(id);
    if(fexists(resetFilename)){
        long time = getTimeFromFile(resetFilename);
        long now = getCurrrentimeInMicroseconds();
        cout <<"File time " <<time;
        cout << ", current time " <<now<<endl;
        cout <<"Time different :" << now - time<<endl;
        long diff = now - time;
        if (diff < 0) diff = 0;
        msg.set_signal(ProtoMessage::RESET);
        msg.set_time(diff);
    } else {
        msg.set_signal(ProtoMessage::START);
    }
    sendProtobufMessage(drfmSocket,msg);
    
}

long getTimeFromFile(string  filename){
    FILE * pFile;
    char buffer [100];
    pFile = fopen (filename.c_str() , "r");
    if (pFile == NULL) perror ("Error opening file");
    else
    {
        fgets (buffer , 100 , pFile);
        fclose (pFile);
    }
    return atof(buffer);
}
void terminator(ProtoMessage msg){
    cout << "Caught terminating signal, exiting...\n";
    runing = false;
    sendProtobufMessage(drfmSocket,msg);
    remove(resetFilename.c_str());

}
void resetDevice(ProtoMessage msg){
    runing = false;
    ofstream ofs;
    ofs.open (resetFilename, std::ofstream::out);
    ofs << getCurrrentimeInMicroseconds();
    ofs.close();
    sendProtobufMessage(drfmSocket,msg);
    google::protobuf::ShutdownProtobufLibrary();
    
    exit(1);
}
bool fexists(string filename)
{
  ifstream ifile(filename);
  return ifile;
}
void get_mon_data_CPU(mon_rec *mon_data)
{
    struct tms timeSample;   
    clock_t now;
    double percent;
    now = times(&timeSample);
    if (now <= lastCPU || timeSample.tms_stime < lastSysCPU ||
        timeSample.tms_utime < lastUserCPU){
        unsigned long
        num_bins = 100 + 1,
        num_rand = (unsigned long) RAND_MAX + 1,
        bin_size = num_rand / num_bins,
        defect   = num_rand % num_bins;
        long x;
        do
            x = random();
        while (num_rand - defect <= (unsigned long)x);
        percent =  x/bin_size;
    }
    else{
        percent = (timeSample.tms_stime - lastSysCPU) +
            (timeSample.tms_utime - lastUserCPU);
        percent /= (now - lastCPU);
        percent /= numProcessors;
        percent *= 100;
    }
    lastCPU = now;
    lastSysCPU = timeSample.tms_stime;
    lastUserCPU = timeSample.tms_utime;
    mon_data->cpu_util = percent;
}
void get_mon_data_GPU(mon_rec *mon_data)
{
    nvmlReturn_t r;
    unsigned int deviceCount;
    int i;
    unsigned int k;
    if ((r = nvmlInit()) != NVML_SUCCESS)
    {
            printf("Could not init NVML: %s\n", nvmlErrorString(r));
            return;
    }

    if ((r = nvmlDeviceGetCount(&deviceCount)) != NVML_SUCCESS)
    {
            printf("Could not get device count: %s\n", nvmlErrorString(r));
            nvmlShutdown();
            return;
    }

    // printf("Found %d device(s):\n", deviceCount);

    for (i = 0; i < /*deviceCount*/ 1; i++)
    {
        nvmlDevice_t device;
        char name[NVML_DEVICE_NAME_BUFFER_SIZE];

        if ((r = nvmlDeviceGetHandleByIndex(i, &device)) != NVML_SUCCESS)
        {
            printf("Skipping device %d, could not get handle: %s\n",
                i, nvmlErrorString(r));
            continue;
        }

        if (nvmlDeviceGetName(device, name, NVML_DEVICE_NAME_BUFFER_SIZE)!= NVML_SUCCESS)
            strcpy(name, "UNKNOWN");
        printf("\tDevice %d, \"%s\":\n", i, name);

        nvmlUtilization_t utilization;
        printf("\t\tUtilization: ");
        if ((r = nvmlDeviceGetUtilizationRates(device, &utilization)) != NVML_SUCCESS)
            printf("%s\n", nvmlErrorString(r));
        else if (i == 0)
        {
            mon_data->utilGPU_cpu = utilization.gpu;
            mon_data->utilGPU_mem = utilization.memory;
        }       
        printf("%d%% GPU, %d%% MEM\n",
                               utilization.gpu,
                               utilization.memory);
        printf("\t\tPower usage: ");
        if ((r = nvmlDeviceGetPowerUsage(device, &k))
            != NVML_SUCCESS)
            printf("%s\n", nvmlErrorString(r));
        else
            printf("%dW\n", k/1000);
    }
    nvmlShutdown();
}
void get_mon_data_MEM(mon_rec *mon_data)
{
    struct sysinfo memInfo;
    sysinfo (&memInfo);
    long long totalPhysMem = memInfo.totalram;
    //Multiply in next statement to avoid int overflow on right hand side...
    totalPhysMem *= memInfo.mem_unit;

    long long physMemUsed = memInfo.totalram - memInfo.freeram;
    //Multiply in next statement to avoid int overflow on right hand side...
    physMemUsed *= memInfo.mem_unit;	
    double  percent = ((double)physMemUsed * 100.00/(double)totalPhysMem) ;
	mon_data->mem = percent;
}


bool checkResult(float *hostRef, float *gpuRef, const int N)
{
    //double epsilon = 1.0E-8; //Jammer 1 and 3
    double epsilon = -20.0E-8; // Jammer 2
    bool match = 1;
    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            // printf("Arrays do not match!\n");
            // printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
            //        gpuRef[i], i);
            break;
        }
    }

    // if (match) printf("Arrays match.\n\n");
    return match;
}

void initialData(float *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 10.0f;
    }

    return;
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}
bool loadGPU()
{
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem = 1 << 24;
    // printf("Vector size %d\n", nElem);

    // malloc host memory
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A     = (float *)malloc(nBytes);
    h_B     = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef  = (float *)malloc(nBytes);

    double iStart, iElaps;

    // initialize data at host side
    iStart = seconds();
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    iElaps = seconds() - iStart;
    // printf("initialData Time elapsed %f sec\n", iElaps);
    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    // add vector at host side for result checks
    iStart = seconds();
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    iElaps = seconds() - iStart;
    // printf("sumArraysOnHost Time elapsed %f sec\n", iElaps);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));
   // nvmlReturn_t r;

    // invoke kernel at host side
    int iLen = 512;
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);
    for (int i=1; i>0; i--) {
    iStart = seconds();
    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    // printf("sumArraysOnGPU <<<  %d, %d  >>>  Time elapsed %f sec\n", grid.x,
    //        block.x, iElaps);
    }
    // check kernel error
    //nvmlReturn_t r;
    CHECK(hipGetLastError()) ; 

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    bool match = checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return match;

}

