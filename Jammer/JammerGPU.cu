#include "Jammer.h"
// #include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/times.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/sysinfo.h>
#include <string.h>
#include <sys/vtimes.h>
#include <time.h>
#include <unistd.h>
#include <sys/stat.h>
#include <ctype.h>
#include <memory.h>
#include <assert.h>

#include </usr/include/nvidia/gdk/nvml.h>

bool Jammer::initialConnection(){
    cout << "Jammer is Started" <<endl;
    return true;
}
Jammer::Jammer() {
    id = 1;
}




void get_mon_data_MEM(mon_rec *mon_data)
{
    struct sysinfo memInfo;
    sysinfo (&memInfo);
    long long totalPhysMem = memInfo.totalram;
    //Multiply in next statement to avoid int overflow on right hand side...
    totalPhysMem *= memInfo.mem_unit;

    long long physMemUsed = memInfo.totalram - memInfo.freeram;
    //Multiply in next statement to avoid int overflow on right hand side...
    physMemUsed *= memInfo.mem_unit;	
    double  percent = ((double)physMemUsed * 100.00/(double)totalPhysMem) ;
	mon_data->mem = percent;
}
void get_mon_data_GPU(mon_rec *mon_data)
{
    nvmlReturn_t r;
    unsigned int deviceCount;
    int i;
    unsigned int k;
    if ((r = nvmlInit()) != NVML_SUCCESS)
    {
            printf("Could not init NVML: %s\n", nvmlErrorString(r));
            return;
    }

    if ((r = nvmlDeviceGetCount(&deviceCount)) != NVML_SUCCESS)
    {
            printf("Could not get device count: %s\n", nvmlErrorString(r));
            nvmlShutdown();
            return;
    }

    printf("Found %d device(s):\n", deviceCount);

    for (i = 0; i < deviceCount; i++)
    {
        nvmlDevice_t device;
        char name[NVML_DEVICE_NAME_BUFFER_SIZE];

        if ((r = nvmlDeviceGetHandleByIndex(i, &device)) != NVML_SUCCESS)
        {
            printf("Skipping device %d, could not get handle: %s\n",
                i, nvmlErrorString(r));
            continue;
        }

        if (nvmlDeviceGetName(device, name, NVML_DEVICE_NAME_BUFFER_SIZE)!= NVML_SUCCESS)
            strcpy(name, "UNKNOWN");
        printf("\tDevice %d, \"%s\":\n", i, name);

        nvmlUtilization_t utilization;
        printf("\t\tUtilization: ");
        if ((r = nvmlDeviceGetUtilizationRates(device, &utilization)) != NVML_SUCCESS)
            printf("%s\n", nvmlErrorString(r));
        else if (i == 0)
        {
            mon_data->utilGPU_cpu = utilization.gpu;
            mon_data->utilGPU_mem = utilization.memory;
        }       
        printf("%d%% GPU, %d%% MEM\n",
                               utilization.gpu,
                               utilization.memory);
        printf("\t\tPower usage: ");
        if ((r = nvmlDeviceGetPowerUsage(device, &k))
            != NVML_SUCCESS)
            printf("%s\n", nvmlErrorString(r));
        else
            printf("%dW\n", k/1000);
    }
    nvmlShutdown();
}
void get_mon_data_CPU(mon_rec *mon_data)
{
  
    static clock_t lastCPU, lastSysCPU, lastUserCPU;
    static int numProcessors;
    FILE* file;
    struct tms timeSample;
    char line[128];
    lastCPU = times(&timeSample);
    lastSysCPU = timeSample.tms_stime;
    lastUserCPU = timeSample.tms_utime;
    file = fopen("/proc/cpuinfo", "r");
    numProcessors = 0;
    while(fgets(line, 128, file) != NULL){
        if (strncmp(line, "processor", 9) == 0) numProcessors++;
    }
    fclose(file);
    clock_t now;
    double percent;
    now = times(&timeSample);
    if (now <= lastCPU || timeSample.tms_stime < lastSysCPU ||
        timeSample.tms_utime < lastUserCPU){
        double M, N;
        M = 10;
        N = 100;
        percent = (rand() / (RAND_MAX / (N-M)));
    } else {
        percent = (timeSample.tms_stime - lastSysCPU) +
            (timeSample.tms_utime - lastUserCPU);
        percent /= (now - lastCPU);
        percent /= numProcessors;
        percent *= 100;
    }
    lastCPU = now;
    lastSysCPU = timeSample.tms_stime;
    lastUserCPU = timeSample.tms_utime;
	mon_data->cpu_util = percent;
}
int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    Jammer jammer;
    jammer.initialConnection();

    get_mon_data_MEM(&mon_data);
    printf ("Main mem %f\n", mon_data.mem);
    get_mon_data_CPU(&mon_data);
    printf ("Main CPU %f\n", mon_data.cpu_util);
    get_mon_data_GPU(&mon_data);
    printf ("mon_data %lf %lf %lf %lf %u \n", mon_data.mem, mon_data.cpu_util, mon_data.utilGPU_cpu, mon_data.utilGPU_mem, mon_data.app); 
    return(0);
}

